#include "hip/hip_runtime.h"
/*
 * kmeans_parallel.c
 *
 *  Created on: 04/06/2013
 *      Author: gustavo
 */
#include <cstdio>
#include <cstdlib>
#include <string>
#include <cassert>
#include <ctime>
#include <cmath>
#include <cfloat>
#include "utils.h"
#include "kmeans.h"
#include "kmeans_parallel.h"
#include <algorithm>
#include "cudaTimer.h"


__global__
void clear_vectors(float *d_runningSumOfExamplesPerCentroid,
		int *d_numberOfExamplePerCentroid, int nCentroids, int nDim, int *d_changedSinceLastIteration )
{
	int myExample = blockIdx.x * blockDim.x + threadIdx.x;

		if (myExample < nCentroids)
		{
			for (int i = 0; i < nDim; i++)
				d_runningSumOfExamplesPerCentroid[myExample * nDim + i] = 0;
			d_numberOfExamplePerCentroid[myExample] = 0;
		}
		*d_changedSinceLastIteration = 0;
}

#define BLOCKSIZEEXAMPLES 30
#define BLOCKSIZECENTROIDS 30
#define DIMENSIONS 11
__global__
void run_kmeans_parallel(float *d_dataX, float *d_centroidPosition,
		int *d_centroidAssignedToExample,
		float *d_runningSumOfExamplesPerCentroid,
		int *d_numberOfExamplePerCentroid, int nExamples, int nCentroids, int nDim, float *d_distanceExampleCentroid) {


	int myExample = blockIdx.x * blockDim.x + threadIdx.x;
	int myCentroid = blockIdx.y * blockDim.y + threadIdx.y;
	if (myExample >= nExamples || myCentroid >= nCentroids)
		return; //out of range
    int i;
//Copy data to shared memory
#ifdef USESHAREDMEMORY
	__shared__ float examples[BLOCKSIZEEXAMPLES][DIMENSIONS];
    __shared__ float centroids[BLOCKSIZECENTROIDS][DIMENSIONS];


    //Note: Only works if all blocks have more than nDim threads in each dimension (x.y)
    if (threadIdx.y < nDim)
    	examples[threadIdx.x][threadIdx.y] = d_dataX[myExample * nDim + threadIdx.y];

    if (threadIdx.x < nDim)
    	centroids[threadIdx.y][threadIdx.x] = d_centroidPosition [myCentroid * nDim + threadIdx.x];

    __syncthreads();
#endif
	float sum = 0;
	float currentVal;

	//Find closest centroid to current example
	float distanceToCurrentCentroid;

	sum = 0;

	for (i = 0; i < nDim; i++) {

#ifdef USESHAREDMEMORY
		//printf("BlockIDx: %d, BlockIDy: %d, ThreadIDx: %d, ThreadIDy: %d; Examples: %f %f; Centroids: %f, %f\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, examples[threadIdx.x][i], d_dataX[myExample * nDim + i], centroids[threadIdx.y][i], d_centroidPosition[myCentroid * nDim + i]);
//		assert(centroids[threadIdx.y][i] == d_centroidPosition[myCentroid * nDim + i]);
//		assert(examples[threadIdx.x][i] == d_dataX[myExample * nDim + i]);


		currentVal = centroids[threadIdx.y][i] - examples[threadIdx.x][i];
#else
		currentVal = d_centroidPosition[myCentroid * nDim + i] - d_dataX[myExample * nDim + i];
#endif

		sum += currentVal * currentVal;
	}
	distanceToCurrentCentroid = sqrt(sum);

	d_distanceExampleCentroid[myExample * nCentroids + myCentroid] = distanceToCurrentCentroid;

}

__global__
void aggregate_centroid_locations(float *d_runningSumOfExamplesPerCentroid,
		int *d_numberOfExamplePerCentroid, int *d_centroidAssignedToExample, float *d_dataX, int nDim, int nExamples, int nCentroids, float *d_distanceExampleCentroid, int *d_changedSinceLastIteration)
{
	int myExample = blockIdx.x * blockDim.x + threadIdx.x;
	if (myExample >= nExamples)
		return;

	float smallestDistanceToCentroid = FLT_MAX;
	int assignedCentroid;
	float distanceToCurrentCentroid;
	int jCentroid;
	for (jCentroid = 0; jCentroid < nCentroids; jCentroid++)
	{
		distanceToCurrentCentroid = d_distanceExampleCentroid[myExample * nCentroids + jCentroid];
		if (distanceToCurrentCentroid < smallestDistanceToCentroid) {
			smallestDistanceToCentroid = distanceToCurrentCentroid;
			assignedCentroid = jCentroid;
		}
	}

	if (d_centroidAssignedToExample[myExample] != assignedCentroid)
		*d_changedSinceLastIteration = 1;

	d_centroidAssignedToExample[myExample] = assignedCentroid;

	int i;
	atomicAdd(&d_numberOfExamplePerCentroid[assignedCentroid], 1);
	for (i = 0; i < nDim; i++) {
		atomicAdd(&d_runningSumOfExamplesPerCentroid[assignedCentroid * nDim + i], d_dataX[myExample * nDim + i]);
	}

}

__global__
void update_centroids(float *d_centroidPosition,
		float *d_runningSumOfExamplesPerCentroid,
		int *d_numberOfExamplePerCentroid, int nCentroids, int nDim) {

	int myExample = blockIdx.x * blockDim.x + threadIdx.x;

	if (myExample < nCentroids)
	{
		int jDim;
		for (jDim = 0; jDim < nDim; jDim++)
			d_centroidPosition[myExample * nDim + jDim] =
					d_runningSumOfExamplesPerCentroid[myExample * nDim
							+ jDim]
							/ d_numberOfExamplePerCentroid[myExample];
	}

}

float* KmeansParallel::run(int nCentroids, int maxIter) {
	this->nCentroids = nCentroids;
	lastExecutionTime = 0;
	AllocateMemoryForCentroidVariables();

	//InitializeCentroids
	(*initializeCentroidsFunction)(dataX, centroidPosition, nCentroids, nDim,
			nExamples, verbose);

	CudaTimer totalTime;

	totalTime.start();
	AllocateMemoryAndCopyVariablesToGPU();

	int blockSize_examplesDim = BLOCKSIZEEXAMPLES;
	int gridSize_examplesDim = nExamples / blockSize_examplesDim + 1;

	int blockSize_centroidsDim = BLOCKSIZECENTROIDS;
	int gridSize_centroidsDim = nCentroids / blockSize_centroidsDim + 1;

    const dim3 blockSize_main (blockSize_examplesDim, blockSize_centroidsDim, 1);
    const dim3 gridSize_main (gridSize_examplesDim, gridSize_centroidsDim, 1);

	int blockSize_centroidsOnly = 32;
	int gridSize_centroidsOnly = nCentroids / blockSize_centroidsOnly + 1;

    const dim3 blockSize_centroids (blockSize_centroidsOnly,1, 1);
    const dim3 gridSizeCentroids(gridSize_centroidsOnly, 1, 1);

	int blockSize_examplesOnly = 32;
	int gridSize_examplesOnly = nExamples / blockSize_examplesOnly + 1;

    const dim3 blockSize_examples(blockSize_examplesOnly,1, 1);
    const dim3 gridSizeExamples(gridSize_examplesOnly, 1, 1);

    changedSinceLastIteration = 1;
    int nIteration = 0;

    float totalTimeInMainKernel = 0;
    float totalTimeInClearVectorsKernel = 0;
    float totalTimeInAggregateCentroidsKernel = 0;
    float totalTimeInUpdateCentroidsKernel = 0;

    CudaTimer cudaTimer;


    while (changedSinceLastIteration &&
    		(nIteration < maxIter || maxIter == -1)) {
    	if (verbose)
    		printf ("Starting iteration %d:\n", nIteration);

    	cudaTimer.start();
		clear_vectors<<<gridSizeCentroids, blockSize_centroids>>> (d_runningSumOfExamplesPerCentroid, d_numberOfExamplePerCentroid,nCentroids, nDim, d_changedSinceLastIteration);
		syncAndCheckErrors();
		totalTimeInClearVectorsKernel += cudaTimer.stop();

		cudaTimer.start();
		run_kmeans_parallel<<<gridSize_main, blockSize_main>>> (d_dataX, d_centroidPosition,
				d_centroidAssignedToExample, d_runningSumOfExamplesPerCentroid, d_numberOfExamplePerCentroid, nExamples, nCentroids, nDim, d_distanceExampleCentroid);
		syncAndCheckErrors();
		totalTimeInMainKernel += cudaTimer.stop();

		cudaTimer.start();
		aggregate_centroid_locations<<<gridSizeExamples, blockSize_examples>>> (d_runningSumOfExamplesPerCentroid, d_numberOfExamplePerCentroid, d_centroidAssignedToExample, d_dataX, nDim, nExamples, nCentroids, d_distanceExampleCentroid, d_changedSinceLastIteration);
		syncAndCheckErrors();
		totalTimeInAggregateCentroidsKernel += cudaTimer.stop();

		cudaTimer.start();
		update_centroids<<<gridSizeCentroids, blockSize_centroidsOnly>>> (d_centroidPosition,
				d_runningSumOfExamplesPerCentroid, d_numberOfExamplePerCentroid,nCentroids, nDim);
		syncAndCheckErrors();
		totalTimeInUpdateCentroidsKernel += cudaTimer.stop();

		CopyCompletionFlagFromGPU();

		if (nIteration == 0)
			changedSinceLastIteration = true;

		nIteration ++;
    }



    CopyResultsFromGPU();

    float totalTimeInKernels = totalTimeInClearVectorsKernel + totalTimeInMainKernel + totalTimeInAggregateCentroidsKernel + totalTimeInUpdateCentroidsKernel;
    
    float total = totalTime.stop();
    lastExecutionTime = total;

    if (verbose)
    {
		printf("Total time: %f ms\n", total);
		printf("Total kernel time: %f ms\n", totalTimeInKernels);
		printf("Time spent for each kernel: \n");
		printf("Clear: %f ms (%.2f%%)\n", totalTimeInClearVectorsKernel, totalTimeInClearVectorsKernel / totalTimeInKernels * 100);
		printf("Main: %f ms (%.2f%%)\n", totalTimeInMainKernel, totalTimeInMainKernel/ totalTimeInKernels *100);
		printf("Aggregate Centroids: %f ms (%.2f%%)\n", totalTimeInAggregateCentroidsKernel, totalTimeInAggregateCentroidsKernel/ totalTimeInKernels *100);
		printf("Update Centroids: %f ms (%.2f%%)\n", totalTimeInUpdateCentroidsKernel, totalTimeInUpdateCentroidsKernel/ totalTimeInKernels *100);
		printf("Centroids: \n");
		int i;
		for (i = 0; i < nCentroids; i++)
			printf("%.17g %.17g %.17g %.17g\n", centroidPosition[i * nDim + 0],
					centroidPosition[i * nDim + 1], centroidPosition[i * nDim + 2],
					centroidPosition[i * nDim + 3]);
		fflush(stdout);

    }

	FreeHostMemory();
	FreeGPUMemory();


	return centroidPosition;
}


KmeansParallel::KmeansParallel(float *data, int nExamples, int nDim,
		bool verbose) {
	this->dataX = data;
	this->nExamples = nExamples;
	this->nDim = nDim;
	this->verbose = verbose;
	this->initializeCentroidsFunction = &InitializeCentroids;
	centroidPosition = 0;
	lastExecutionTime = 0;
}

KmeansParallel::~KmeansParallel() {

}

float KmeansParallel::getLastRunningTime() {
	return lastExecutionTime;
}

void KmeansParallel::setInitializeCentroidsFunction(initFunction fun) {
	initializeCentroidsFunction = fun;
}

int KmeansParallel::FindClosestCentroidsAndCheckForChanges() {
	//Find closest centroids
	int changedFromLastIteration = 0;
	for (int iExample = 0; iExample < nExamples; iExample++) {
		int closestCentroid = GetClosestCentroid(iExample);
		if (closestCentroid != centroidAssignedToExample[iExample])
			changedFromLastIteration = 1;
		centroidAssignedToExample[iExample] = closestCentroid;
	}
	return changedFromLastIteration;
}

void KmeansParallel::syncAndCheckErrors() {
	hipDeviceSynchronize();
	checkCudaErrors (hipGetLastError());}



void KmeansParallel::AllocateMemoryForCentroidVariables() {
	//Allocate memory for centroid variables
	centroidPosition = (float*) malloc(sizeof(float) * (nCentroids * nDim));
	centroidAssignedToExample = (int*) malloc(sizeof(int) * nExamples);
	runningSumOfExamplesPerCentroid = (float*) malloc(
			sizeof(float) * (nCentroids * nDim));
	numberOfExamplePerCentroid = (int*) ((malloc(sizeof(int) * nCentroids)));
}

void KmeansParallel::AllocateMemoryAndCopyVariablesToGPU() {
	checkCudaErrors(hipMalloc(&d_dataX, sizeof(float) * nExamples * nDim ));
	checkCudaErrors(
			hipMalloc(&d_centroidPosition, sizeof(float) * (nCentroids * nDim)));
	checkCudaErrors(
			hipMalloc(&d_centroidAssignedToExample, sizeof(int) * nExamples));
	checkCudaErrors(
			hipMalloc(&d_runningSumOfExamplesPerCentroid, sizeof(float) * (nCentroids * nDim)));
	checkCudaErrors(
			hipMalloc(&d_numberOfExamplePerCentroid, sizeof(int) * nCentroids ));
	checkCudaErrors(
				hipMalloc(&d_changedSinceLastIteration, sizeof(int) ));
	checkCudaErrors(
					hipMalloc(&d_distanceExampleCentroid, sizeof(float) * nExamples * nCentroids ));


	checkCudaErrors(
			hipMemcpy(d_centroidPosition, centroidPosition, sizeof(float) * (nCentroids * nDim), hipMemcpyHostToDevice));
	checkCudaErrors(
			hipMemcpy(d_dataX, dataX, sizeof(float) * (nExamples * nDim), hipMemcpyHostToDevice));
}

void KmeansParallel::FreeHostMemory() {
	free(centroidPosition);
	free(centroidAssignedToExample);
	free(runningSumOfExamplesPerCentroid);
	free(numberOfExamplePerCentroid);
}

void KmeansParallel::FreeGPUMemory() {
	checkCudaErrors(hipFree(d_dataX));
	checkCudaErrors(hipFree(d_centroidPosition));
	checkCudaErrors(hipFree(d_centroidAssignedToExample));
	checkCudaErrors(hipFree(d_runningSumOfExamplesPerCentroid));
	checkCudaErrors(hipFree(d_numberOfExamplePerCentroid));
	checkCudaErrors(hipFree(d_changedSinceLastIteration));
	checkCudaErrors(hipFree(d_distanceExampleCentroid));
}

void KmeansParallel::CopyCompletionFlagFromGPU(){
	checkCudaErrors(hipMemcpy(&changedSinceLastIteration, d_changedSinceLastIteration, sizeof(int) , hipMemcpyDeviceToHost));
}

void KmeansParallel::CopyResultsFromGPU(){
	checkCudaErrors(hipMemcpy(centroidPosition, d_centroidPosition, sizeof(float) * (nCentroids * nDim), hipMemcpyDeviceToHost));
}

void KmeansParallel::ClearIntArray(int* vector, int size) {
	int i;
	for (i = 0; i < size; i++)
		vector[i] = 0;
}

void KmeansParallel::ClearfloatArray(float* vector, int size) {
	int i;
	for (i = 0; i < size; i++)
		vector[i] = 0.0;
}

void KmeansParallel::InitializeCentroids(float *dataX, float *centroidPosition,
		int nCentroids, int nDim, int nExamples, bool verbose) {
	//Initialize centroids with K random examples (Forgy's method)

    int *randomVector;
    int i,j;

    randomVector = (int*) malloc (sizeof(int) * nExamples);
    for (int i =0; i< nExamples;i++)
    	randomVector[i] = i;

    std::random_shuffle(randomVector, randomVector+nExamples);

    if(verbose)
    	printf("Centroids initialized with examples: ");
	int selectedExample;
	for (i = 0; i < nCentroids; i++) {
		selectedExample = randomVector[i];
		if(verbose)
			printf("%d ", selectedExample);
		for (j = 0; j < nDim; j++)
					centroidPosition[i * nDim + j] = dataX[selectedExample * nDim + j];
	}
	if(verbose)
		printf("\n");
	free(randomVector);
}

float KmeansParallel::CalculateDistance(float *dataX, float *centroidPosition,
		int iExample, int jCentroid) {
	//calculate the distance between a data point and a centroid
	int i;
	float sum = 0;
	float currentVal;
	for (i = 0; i < nDim; i++) {
		currentVal = centroidPosition[jCentroid * nDim + i]
				- dataX[iExample * nDim + i];
		sum += currentVal * currentVal;
	}
	return sqrt(sum);
}

int KmeansParallel::GetClosestCentroid(int iExample) {
	//Find the centroid closest to a data point
	float distanceToCurrentCentroid;
	float smallestDistanceToCentroid = DBL_MAX;
	int assignedCentroid = -1;
	int jCentroid;
	for (jCentroid = 0; jCentroid < nCentroids; jCentroid++) {
		distanceToCurrentCentroid = CalculateDistance(dataX, centroidPosition,
				iExample, jCentroid);
		if (distanceToCurrentCentroid < smallestDistanceToCentroid) {
			smallestDistanceToCentroid = distanceToCurrentCentroid;
			assignedCentroid = jCentroid;
		}
	}

	assert(assignedCentroid != -1);
	return assignedCentroid;
}
