#include "hip/hip_runtime.h"
/*
 * kmeans_parallel.c
 *
 *  Created on: 04/06/2013
 *      Author: gustavo
 */

#include <cstdio>
#include <cstdlib>
#include <string>
#include <cassert>
#include <ctime>
#include <cmath>
#include <cfloat>
#include "iris_data.h"
//#include "load_cifar10_data.h"
#include "kmeans.h"
#include "kmeans_parallel.h"

KmeansParallel::KmeansParallel(float *data, int nExamples, int nDim,
		bool verbose) {
	this->dataX = data;
	this->nExamples = nExamples;
	this->nDim = nDim;
	this->verbose = verbose;
	this->initializeCentroidsFunction = &InitializeCentroids;
}

void KmeansParallel::setInitializeCentroidsFunction(initFunction fun) {
	initializeCentroidsFunction = fun;
}

int KmeansParallel::FindClosestCentroidsAndCheckForChanges() {
	//Find closest centroids
	int changedFromLastIteration = 0;
	for (int iExample = 0; iExample < nExamples; iExample++) {
		int closestCentroid = GetClosestCentroid(iExample);
		if (closestCentroid != centroidAssignedToExample[iExample])
			changedFromLastIteration = 1;
		centroidAssignedToExample[iExample] = closestCentroid;
	}
	return changedFromLastIteration;
}


float* KmeansParallel::run(int nCentroids) {
	this->nCentroids = nCentroids;
	int iExample, changedFromLastIteration;
	AllocateMemoryForCentroidVariables();

	//InitializeCentroids
	(*initializeCentroidsFunction)(dataX, centroidPosition, nCentroids, nDim, nExamples);

	changedFromLastIteration = 1;
	int nIteration = 0;
	while (changedFromLastIteration) {
		nIteration++;
		if (this->verbose)
			printf("Starting iteration %d\n", nIteration);

		changedFromLastIteration = FindClosestCentroidsAndCheckForChanges();

		//Update centroid location
		ClearIntArray(numberOfExamplePerCentroid, nCentroids);
		ClearfloatArray(runningSumOfExamplesPerCentroid, nCentroids * nDim);

		int currentCentroid;
		for (iExample = 0; iExample < nExamples; iExample++) {
			currentCentroid = centroidAssignedToExample[iExample];
			numberOfExamplePerCentroid[currentCentroid]++;
			int jDim;
			for (jDim = 0; jDim < nDim; jDim++)
				runningSumOfExamplesPerCentroid[currentCentroid * nDim + jDim] +=
						dataX[iExample * nDim + jDim];
		}
		for (currentCentroid = 0; currentCentroid < nCentroids;
				currentCentroid++) {
			int jDim;
			for (jDim = 0; jDim < nDim; jDim++)
				centroidPosition[currentCentroid * nDim + jDim] =
						runningSumOfExamplesPerCentroid[currentCentroid * nDim
								+ jDim]
								/ numberOfExamplePerCentroid[currentCentroid];
		}

	}
	printf("done\n");
	printf("Centroids: \n");
	int i;
	for (i = 0; i < nCentroids; i++)
		printf("%.17g %.17g %.17g %.17g\n", centroidPosition[i * nDim + 0],
				centroidPosition[i * nDim + 1], centroidPosition[i * nDim + 2],
				centroidPosition[i * nDim + 3]);
	fflush(stdout);
	return centroidPosition;
}


void KmeansParallel::AllocateMemoryForCentroidVariables() {
	//Allocate memory for centroid variables
	centroidPosition = (float*) malloc(sizeof(float) * (nCentroids * nDim));
	centroidAssignedToExample = (int*) malloc(sizeof(int) * nExamples);
	runningSumOfExamplesPerCentroid = (float*) malloc(
			sizeof(float) * (nCentroids * nDim));
	numberOfExamplePerCentroid = (int*) ((malloc(sizeof(int) * nCentroids)));
}

void KmeansParallel::AllocateMemoryAndCopyVariablesToGPU() {

}


void KmeansParallel::ClearIntArray(int* vector, int size) {
	int i;
	for (i = 0; i < size; i++)
		vector[i] = 0;
}


void KmeansParallel::ClearfloatArray(float* vector, int size) {
	int i;
	for (i = 0; i < size; i++)
		vector[i] = 0.0;
}


void KmeansParallel::InitializeCentroids(float *dataX, float *centroidPosition,
		int nCentroids, int nDim, int nExamples) {
	//Initialize centroids with K random examples (Forgy's method)
	int i;
	printf("Centroids initialized with examples: ");
	int selectedExample;
	for (i = 0; i < nCentroids; i++) {
		selectedExample = rand() % nExamples;
		printf("%d ", selectedExample);
		centroidPosition[i * nDim + 0] = dataX[selectedExample * nDim + 0];
		centroidPosition[i * nDim + 1] = dataX[selectedExample * nDim + 1];
		centroidPosition[i * nDim + 2] = dataX[selectedExample * nDim + 2];
		centroidPosition[i * nDim + 3] = dataX[selectedExample * nDim + 3];
	}
	printf("\n");

}


float KmeansParallel::CalculateDistance(float *dataX, float *centroidPosition, int iExample,
		int jCentroid) {
	//calculate the distance between a data point and a centroid
	int i;
	float sum = 0;
	float currentVal;
	for (i = 0; i < nDim; i++) {
		currentVal = centroidPosition[jCentroid * nDim + i]
				- dataX[iExample * nDim + i];
		sum += currentVal * currentVal;
	}
	return sqrt(sum);
}


int KmeansParallel::GetClosestCentroid(int iExample) {
	//Find the centroid closest to a data point
	float distanceToCurrentCentroid;
	float smallestDistanceToCentroid = DBL_MAX;
	int assignedCentroid = -1;
	int jCentroid;
	for (jCentroid = 0; jCentroid < nCentroids; jCentroid++) {
		distanceToCurrentCentroid = CalculateDistance(dataX, centroidPosition,
				iExample, jCentroid);
		if (distanceToCurrentCentroid < smallestDistanceToCentroid) {
			smallestDistanceToCentroid = distanceToCurrentCentroid;
			assignedCentroid = jCentroid;
		}
	}

	assert(assignedCentroid != -1);
	return assignedCentroid;
}


void KmeansParallel::CompareTestResultsAgainstBaseline(float *centroidPosition) {
	int nCentroids = 3;
	float baseline[] = { 5.0059999999999993, 3.4180000000000006, 1.464,
			0.24399999999999991, 6.8538461538461526, 3.0769230769230766,
			5.7153846153846146, 2.0538461538461532, 5.8836065573770497,
			2.7409836065573772, 4.3885245901639349, 1.4344262295081966 };
	int i;
	float maxError = 1e-5;
	float error = 0;
	for (i = 0; i < nCentroids * nDim; i++)
		error += fabs(centroidPosition[i] - baseline[i]);

	assert(error < maxError);
	printf("OK!! Error agains baseline below threshold: %lf\n", error);
}


