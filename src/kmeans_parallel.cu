#include "hip/hip_runtime.h"
/*
 * kmeans_parallel.c
 *
 *  Created on: 04/06/2013
 *      Author: gustavo
 */
#include <cstdio>
#include <cstdlib>
#include <string>
#include <cassert>
#include <ctime>
#include <cmath>
#include <cfloat>
#include "utils.h"
#include "kmeans.h"
#include "kmeans_parallel.h"


__global__
void run_kmeans_parallel(float *d_dataX, float *d_centroidPosition,
		int *d_centroidAssignedToExample,
		float *d_runningSumOfExamplesPerCentroid,
		int *d_numberOfExamplePerCentroid, int nExamples, int nCentroids, int nDim, int *d_changedSinceLastIteration) {

	int myExample = blockIdx.x * blockDim.x + threadIdx.x;
	if (myExample >= nExamples)
		return; //out of range

	int i;
	float sum = 0;
	float currentVal;

	//Find closest centroid to current example
	float distanceToCurrentCentroid;
	float smallestDistanceToCentroid = FLT_MAX;
	int assignedCentroid = -1;
	int jCentroid;
	for (jCentroid = 0; jCentroid < nCentroids; jCentroid++) {
		sum = 0;

		for (i = 0; i < nDim; i++) {
			currentVal = d_centroidPosition[jCentroid * nDim + i]
					- d_dataX[myExample * nDim + i];
			sum += currentVal * currentVal;
		}
		distanceToCurrentCentroid = sqrt(sum);

		if (distanceToCurrentCentroid < smallestDistanceToCentroid) {
			smallestDistanceToCentroid = distanceToCurrentCentroid;
			assignedCentroid = jCentroid;
		}
	}
	if (d_centroidAssignedToExample[myExample] != assignedCentroid)
		*d_changedSinceLastIteration = 1;

	d_centroidAssignedToExample[myExample] = assignedCentroid;

	atomicAdd(&d_numberOfExamplePerCentroid[assignedCentroid], 1);
	for (i = 0; i < nDim; i++) {
		atomicAdd(&d_runningSumOfExamplesPerCentroid[assignedCentroid], d_dataX[myExample * nDim + i]);
	}

	//synchronize threads
	__syncthreads();

	if (myExample < nCentroids)
	{
		int jDim;
		for (jDim = 0; jDim < nDim; jDim++)
			d_centroidPosition[myExample * nDim + jDim] =
					d_runningSumOfExamplesPerCentroid[myExample * nDim
							+ jDim]
							/ d_numberOfExamplePerCentroid[myExample];
	}



}

/*
	changedFromLastIteration = 1;
	int nIteration = 0;
	while (changedFromLastIteration) {
		nIteration++;
		if (this->verbose)
			printf("Starting iteration %d\n", nIteration);


		//Update centroid location
		ClearIntArray(numberOfExamplePerCentroid, nCentroids);
		ClearfloatArray(runningSumOfExamplesPerCentroid, nCentroids * nDim);

		int currentCentroid;
		for (iExample = 0; iExample < nExamples; iExample++) {
			currentCentroid = centroidAssignedToExample[iExample];
			numberOfExamplePerCentroid[currentCentroid]++;
			int jDim;
			for (jDim = 0; jDim < nDim; jDim++)
				runningSumOfExamplesPerCentroid[currentCentroid * nDim + jDim] +=
						dataX[iExample * nDim + jDim];
		}
		for (currentCentroid = 0; currentCentroid < nCentroids;
				currentCentroid++) {
			int jDim;
			for (jDim = 0; jDim < nDim; jDim++)
				centroidPosition[currentCentroid * nDim + jDim] =
						runningSumOfExamplesPerCentroid[currentCentroid * nDim
								+ jDim]
								/ numberOfExamplePerCentroid[currentCentroid];
		}

	}*/


KmeansParallel::KmeansParallel(float *data, int nExamples, int nDim,
		bool verbose) {
	this->dataX = data;
	this->nExamples = nExamples;
	this->nDim = nDim;
	this->verbose = verbose;
	this->initializeCentroidsFunction = &InitializeCentroids;
}

void KmeansParallel::setInitializeCentroidsFunction(initFunction fun) {
	initializeCentroidsFunction = fun;
}

int KmeansParallel::FindClosestCentroidsAndCheckForChanges() {
	//Find closest centroids
	int changedFromLastIteration = 0;
	for (int iExample = 0; iExample < nExamples; iExample++) {
		int closestCentroid = GetClosestCentroid(iExample);
		if (closestCentroid != centroidAssignedToExample[iExample])
			changedFromLastIteration = 1;
		centroidAssignedToExample[iExample] = closestCentroid;
	}
	return changedFromLastIteration;
}

float* KmeansParallel::run(int nCentroids) {
	this->nCentroids = nCentroids;
	AllocateMemoryForCentroidVariables();

	//InitializeCentroids
	(*initializeCentroidsFunction)(dataX, centroidPosition, nCentroids, nDim,
			nExamples);

	AllocateMemoryAndCopyVariablesToGPU();

	//1 thread per example
	int blockSize_1d = 512;
	int gridSize_1d = nExamples / blockSize_1d + 1;

	printf ("nExamples: %d; blockSize: %d; gridSize:%d\n", nExamples, blockSize_1d, gridSize_1d);

    const dim3 blockSize (blockSize_1d, 1, 1);
    const dim3 gridSize(gridSize_1d, 1, 1);



	run_kmeans_parallel<<<gridSize, blockSize>>> (d_dataX, d_centroidPosition,
			d_centroidAssignedToExample, d_runningSumOfExamplesPerCentroid, d_numberOfExamplePerCentroid, nExamples, nCentroids, nDim, d_changedSinceLastIteration);

	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	ClearfloatArray(centroidPosition, nCentroids * nDim);

	CopyResultsFromGPU();

	printf("done\n");
	printf("Centroids: \n");
	int i;
	for (i = 0; i < nCentroids; i++)
		printf("%.17g %.17g %.17g %.17g\n", centroidPosition[i * nDim + 0],
				centroidPosition[i * nDim + 1], centroidPosition[i * nDim + 2],
				centroidPosition[i * nDim + 3]);
	fflush(stdout);
	return centroidPosition;
}

void KmeansParallel::AllocateMemoryForCentroidVariables() {
	//Allocate memory for centroid variables
	centroidPosition = (float*) malloc(sizeof(float) * (nCentroids * nDim));
	centroidAssignedToExample = (int*) malloc(sizeof(int) * nExamples);
	runningSumOfExamplesPerCentroid = (float*) malloc(
			sizeof(float) * (nCentroids * nDim));
	numberOfExamplePerCentroid = (int*) ((malloc(sizeof(int) * nCentroids)));
}

void KmeansParallel::AllocateMemoryAndCopyVariablesToGPU() {
	checkCudaErrors(hipMalloc(&d_dataX, sizeof(float) * nExamples * nDim ));
	checkCudaErrors(
			hipMalloc(&d_centroidPosition, sizeof(float) * (nCentroids * nDim)));
	checkCudaErrors(
			hipMalloc(&d_centroidAssignedToExample, sizeof(int) * nExamples));
	checkCudaErrors(
			hipMalloc(&d_runningSumOfExamplesPerCentroid, sizeof(float) * (nCentroids * nDim)));
	checkCudaErrors(
			hipMalloc(&d_numberOfExamplePerCentroid, sizeof(int) * nCentroids ));
	checkCudaErrors(
				hipMalloc(&d_changedSinceLastIteration, sizeof(int) ));

	checkCudaErrors(
			hipMemcpy(d_centroidPosition, centroidPosition, sizeof(float) * (nCentroids * nDim), hipMemcpyHostToDevice));
	checkCudaErrors(
			hipMemcpy(d_dataX, dataX, sizeof(float) * (nExamples * nDim), hipMemcpyHostToDevice));
}

void KmeansParallel::CopyResultsFromGPU(){
	checkCudaErrors(hipMemcpy(centroidPosition, d_centroidPosition, sizeof(float) * (nCentroids * nDim), hipMemcpyDeviceToHost));
}

void KmeansParallel::ClearIntArray(int* vector, int size) {
	int i;
	for (i = 0; i < size; i++)
		vector[i] = 0;
}

void KmeansParallel::ClearfloatArray(float* vector, int size) {
	int i;
	for (i = 0; i < size; i++)
		vector[i] = 0.0;
}

void KmeansParallel::InitializeCentroids(float *dataX, float *centroidPosition,
		int nCentroids, int nDim, int nExamples) {
	//Initialize centroids with K random examples (Forgy's method)
	int i;
	printf("Centroids initialized with examples: ");
	int selectedExample;
	for (i = 0; i < nCentroids; i++) {
		selectedExample = rand() % nExamples;
		printf("%d ", selectedExample);
		centroidPosition[i * nDim + 0] = dataX[selectedExample * nDim + 0];
		centroidPosition[i * nDim + 1] = dataX[selectedExample * nDim + 1];
		centroidPosition[i * nDim + 2] = dataX[selectedExample * nDim + 2];
		centroidPosition[i * nDim + 3] = dataX[selectedExample * nDim + 3];
	}
	printf("\n");

}

float KmeansParallel::CalculateDistance(float *dataX, float *centroidPosition,
		int iExample, int jCentroid) {
	//calculate the distance between a data point and a centroid
	int i;
	float sum = 0;
	float currentVal;
	for (i = 0; i < nDim; i++) {
		currentVal = centroidPosition[jCentroid * nDim + i]
				- dataX[iExample * nDim + i];
		sum += currentVal * currentVal;
	}
	return sqrt(sum);
}

int KmeansParallel::GetClosestCentroid(int iExample) {
	//Find the centroid closest to a data point
	float distanceToCurrentCentroid;
	float smallestDistanceToCentroid = DBL_MAX;
	int assignedCentroid = -1;
	int jCentroid;
	for (jCentroid = 0; jCentroid < nCentroids; jCentroid++) {
		distanceToCurrentCentroid = CalculateDistance(dataX, centroidPosition,
				iExample, jCentroid);
		if (distanceToCurrentCentroid < smallestDistanceToCentroid) {
			smallestDistanceToCentroid = distanceToCurrentCentroid;
			assignedCentroid = jCentroid;
		}
	}

	assert(assignedCentroid != -1);
	return assignedCentroid;
}

void KmeansParallel::CompareTestResultsAgainstBaseline(
		float *centroidPosition) {
	int nCentroids = 3;
	float baseline[] = { 5.0059999999999993, 3.4180000000000006, 1.464,
			0.24399999999999991, 6.8538461538461526, 3.0769230769230766,
			5.7153846153846146, 2.0538461538461532, 5.8836065573770497,
			2.7409836065573772, 4.3885245901639349, 1.4344262295081966 };
	int i;
	float maxError = 1e-5;
	float error = 0;
	for (i = 0; i < nCentroids * nDim; i++)
		error += fabs(centroidPosition[i] - baseline[i]);

	assert(error < maxError);
	printf("OK!! Error agains baseline below threshold: %lf\n", error);
}

