#include "hip/hip_runtime.h"
/*
 * kmeans_parallel.c
 *
 *  Created on: 04/06/2013
 *      Author: gustavo
 */
#include <cstdio>
#include <cstdlib>
#include <string>
#include <cassert>
#include <ctime>
#include <cmath>
#include <cfloat>
#include "utils.h"
#include "kmeans.h"
#include "kmeans_parallel.h"
#include <algorithm>


__global__
void clear_vectors(float *d_runningSumOfExamplesPerCentroid,
		int *d_numberOfExamplePerCentroid, int nCentroids, int nDim, int *d_changedSinceLastIteration )
{
	int myExample = blockIdx.x * blockDim.x + threadIdx.x;

		if (myExample < nCentroids)
		{
			for (int i = 0; i < nDim; i++)
				d_runningSumOfExamplesPerCentroid[myExample * nDim + i] = 0;
			d_numberOfExamplePerCentroid[myExample] = 0;
		}
		*d_changedSinceLastIteration = 0;
}

__global__
void run_kmeans_parallel(float *d_dataX, float *d_centroidPosition,
		int *d_centroidAssignedToExample,
		float *d_runningSumOfExamplesPerCentroid,
		int *d_numberOfExamplePerCentroid, int nExamples, int nCentroids, int nDim, int *d_changedSinceLastIteration) {

	int myExample = blockIdx.x * blockDim.x + threadIdx.x;
	if (myExample >= nExamples)
		return; //out of range

	int i;
	float sum = 0;
	float currentVal;

	//Find closest centroid to current example
	float distanceToCurrentCentroid;
	float smallestDistanceToCentroid = FLT_MAX;
	int assignedCentroid = -1;
	int jCentroid;
	for (jCentroid = 0; jCentroid < nCentroids; jCentroid++) {
		sum = 0;

		for (i = 0; i < nDim; i++) {
			currentVal = d_centroidPosition[jCentroid * nDim + i]
					- d_dataX[myExample * nDim + i];
			sum += currentVal * currentVal;
		}
		distanceToCurrentCentroid = sqrt(sum);

		if (distanceToCurrentCentroid < smallestDistanceToCentroid) {
			smallestDistanceToCentroid = distanceToCurrentCentroid;
			assignedCentroid = jCentroid;
		}
	}
	if (d_centroidAssignedToExample[myExample] != assignedCentroid)
		*d_changedSinceLastIteration = 1;

	d_centroidAssignedToExample[myExample] = assignedCentroid;

	atomicAdd(&d_numberOfExamplePerCentroid[assignedCentroid], 1);
	for (i = 0; i < nDim; i++) {
		atomicAdd(&d_runningSumOfExamplesPerCentroid[assignedCentroid * nDim + i], d_dataX[myExample * nDim + i]);
	}

}

__global__
void update_centroids(float *d_centroidPosition,
		float *d_runningSumOfExamplesPerCentroid,
		int *d_numberOfExamplePerCentroid, int nCentroids, int nDim) {

	int myExample = blockIdx.x * blockDim.x + threadIdx.x;

	if (myExample < nCentroids)
	{
		int jDim;
		for (jDim = 0; jDim < nDim; jDim++)
			d_centroidPosition[myExample * nDim + jDim] =
					d_runningSumOfExamplesPerCentroid[myExample * nDim
							+ jDim]
							/ d_numberOfExamplePerCentroid[myExample];
	}

}


KmeansParallel::KmeansParallel(float *data, int nExamples, int nDim,
		bool verbose) {
	this->dataX = data;
	this->nExamples = nExamples;
	this->nDim = nDim;
	this->verbose = verbose;
	this->initializeCentroidsFunction = &InitializeCentroids;
	centroidPosition = 0;
}

KmeansParallel::~KmeansParallel() {
	if (centroidPosition) //if executed
	{
		FreeHostMemory();
		FreeGPUMemory();
	}

}

void KmeansParallel::setInitializeCentroidsFunction(initFunction fun) {
	initializeCentroidsFunction = fun;
}

int KmeansParallel::FindClosestCentroidsAndCheckForChanges() {
	//Find closest centroids
	int changedFromLastIteration = 0;
	for (int iExample = 0; iExample < nExamples; iExample++) {
		int closestCentroid = GetClosestCentroid(iExample);
		if (closestCentroid != centroidAssignedToExample[iExample])
			changedFromLastIteration = 1;
		centroidAssignedToExample[iExample] = closestCentroid;
	}
	return changedFromLastIteration;
}

float* KmeansParallel::run(int nCentroids, int maxIter) {
	this->nCentroids = nCentroids;
	AllocateMemoryForCentroidVariables();

	//InitializeCentroids
	(*initializeCentroidsFunction)(dataX, centroidPosition, nCentroids, nDim,
			nExamples);

	AllocateMemoryAndCopyVariablesToGPU();

	//1 thread per example
	int blockSize_1d = 512;
	int gridSize_1d = nExamples / blockSize_1d + 1;

	int gridSizeCentroids_1d = nCentroids / blockSize_1d + 1;

	printf ("nExamples: %d; blockSize: %d; gridSize:%d\n", nExamples, blockSize_1d, gridSize_1d);

    const dim3 blockSize (blockSize_1d, 1, 1);
    const dim3 gridSize(gridSize_1d, 1, 1);

    const dim3 gridSizeCentroids(gridSizeCentroids_1d, 1, 1);

    changedSinceLastIteration = 1;
    int nIteration = 0;

    while (changedSinceLastIteration &&
    		(nIteration < maxIter || maxIter == -1)) {
    	printf ("Starting iteration %d:\n", nIteration);

		clear_vectors<<<gridSizeCentroids, blockSize>>> (d_runningSumOfExamplesPerCentroid, d_numberOfExamplePerCentroid,nCentroids, nDim, d_changedSinceLastIteration);


		hipDeviceSynchronize();
		checkCudaErrors(hipGetLastError());

		run_kmeans_parallel<<<gridSize, blockSize>>> (d_dataX, d_centroidPosition,
				d_centroidAssignedToExample, d_runningSumOfExamplesPerCentroid, d_numberOfExamplePerCentroid, nExamples, nCentroids, nDim, d_changedSinceLastIteration);

		hipDeviceSynchronize();
		checkCudaErrors(hipGetLastError());

		update_centroids<<<gridSizeCentroids, blockSize>>> (d_centroidPosition,
				d_runningSumOfExamplesPerCentroid, d_numberOfExamplePerCentroid,nCentroids, nDim);

		hipDeviceSynchronize();
		checkCudaErrors(hipGetLastError());

		CopyCompletionFlagFromGPU();

		if (nIteration == 0)
			changedSinceLastIteration = true;

		nIteration ++;
    }
    CopyResultsFromGPU();

	printf("done\n");
	printf("Centroids: \n");
	int i;
	for (i = 0; i < nCentroids; i++)
		printf("%.17g %.17g %.17g %.17g\n", centroidPosition[i * nDim + 0],
				centroidPosition[i * nDim + 1], centroidPosition[i * nDim + 2],
				centroidPosition[i * nDim + 3]);
	fflush(stdout);
	return centroidPosition;
}

void KmeansParallel::AllocateMemoryForCentroidVariables() {
	//Allocate memory for centroid variables
	centroidPosition = (float*) malloc(sizeof(float) * (nCentroids * nDim));
	centroidAssignedToExample = (int*) malloc(sizeof(int) * nExamples);
	runningSumOfExamplesPerCentroid = (float*) malloc(
			sizeof(float) * (nCentroids * nDim));
	numberOfExamplePerCentroid = (int*) ((malloc(sizeof(int) * nCentroids)));
}

void KmeansParallel::AllocateMemoryAndCopyVariablesToGPU() {

	printf("total_memory = %lu\n", sizeof(float) * nExamples * nDim + sizeof(float) * (nCentroids * nDim) + sizeof(int) * nExamples +
			sizeof(float) * (nCentroids * nDim) + sizeof(int) * nCentroids + sizeof(int));



	checkCudaErrors(hipMalloc(&d_dataX, sizeof(float) * nExamples * nDim ));
	checkCudaErrors(
			hipMalloc(&d_centroidPosition, sizeof(float) * (nCentroids * nDim)));
	checkCudaErrors(
			hipMalloc(&d_centroidAssignedToExample, sizeof(int) * nExamples));
	checkCudaErrors(
			hipMalloc(&d_runningSumOfExamplesPerCentroid, sizeof(float) * (nCentroids * nDim)));
	checkCudaErrors(
			hipMalloc(&d_numberOfExamplePerCentroid, sizeof(int) * nCentroids ));
	checkCudaErrors(
				hipMalloc(&d_changedSinceLastIteration, sizeof(int) ));

	checkCudaErrors(
			hipMemcpy(d_centroidPosition, centroidPosition, sizeof(float) * (nCentroids * nDim), hipMemcpyHostToDevice));
	checkCudaErrors(
			hipMemcpy(d_dataX, dataX, sizeof(float) * (nExamples * nDim), hipMemcpyHostToDevice));
}

void KmeansParallel::FreeHostMemory() {
	free(centroidPosition);
	free(centroidAssignedToExample);
	free(runningSumOfExamplesPerCentroid);
	free(numberOfExamplePerCentroid);
}

void KmeansParallel::FreeGPUMemory() {
	checkCudaErrors(hipFree(d_dataX));
	checkCudaErrors(hipFree(d_centroidPosition));
	checkCudaErrors(hipFree(d_centroidAssignedToExample));
	checkCudaErrors(hipFree(d_runningSumOfExamplesPerCentroid));
	checkCudaErrors(hipFree(d_numberOfExamplePerCentroid));
	checkCudaErrors(hipFree(d_changedSinceLastIteration));
}

void KmeansParallel::CopyCompletionFlagFromGPU(){
	checkCudaErrors(hipMemcpy(&changedSinceLastIteration, d_changedSinceLastIteration, sizeof(int) , hipMemcpyDeviceToHost));
}

void KmeansParallel::CopyResultsFromGPU(){
	checkCudaErrors(hipMemcpy(centroidPosition, d_centroidPosition, sizeof(float) * (nCentroids * nDim), hipMemcpyDeviceToHost));
}

void KmeansParallel::ClearIntArray(int* vector, int size) {
	int i;
	for (i = 0; i < size; i++)
		vector[i] = 0;
}

void KmeansParallel::ClearfloatArray(float* vector, int size) {
	int i;
	for (i = 0; i < size; i++)
		vector[i] = 0.0;
}

void KmeansParallel::InitializeCentroids(float *dataX, float *centroidPosition,
		int nCentroids, int nDim, int nExamples) {
	//Initialize centroids with K random examples (Forgy's method)

    int *randomVector;
    int i;

    randomVector = (int*) malloc (sizeof(int) * nExamples);
    for (int i =0; i< nExamples;i++)
    	randomVector[i] = i;

    std::random_shuffle(randomVector, randomVector+100);

	printf("Centroids initialized with examples: ");
	int selectedExample;
	for (i = 0; i < nCentroids; i++) {
		selectedExample = randomVector[i];
		printf("%d ", selectedExample);
		centroidPosition[i * nDim + 0] = dataX[selectedExample * nDim + 0];
		centroidPosition[i * nDim + 1] = dataX[selectedExample * nDim + 1];
		centroidPosition[i * nDim + 2] = dataX[selectedExample * nDim + 2];
		centroidPosition[i * nDim + 3] = dataX[selectedExample * nDim + 3];
	}
	printf("\n");
	free(randomVector);
}

float KmeansParallel::CalculateDistance(float *dataX, float *centroidPosition,
		int iExample, int jCentroid) {
	//calculate the distance between a data point and a centroid
	int i;
	float sum = 0;
	float currentVal;
	for (i = 0; i < nDim; i++) {
		currentVal = centroidPosition[jCentroid * nDim + i]
				- dataX[iExample * nDim + i];
		sum += currentVal * currentVal;
	}
	return sqrt(sum);
}

int KmeansParallel::GetClosestCentroid(int iExample) {
	//Find the centroid closest to a data point
	float distanceToCurrentCentroid;
	float smallestDistanceToCentroid = DBL_MAX;
	int assignedCentroid = -1;
	int jCentroid;
	for (jCentroid = 0; jCentroid < nCentroids; jCentroid++) {
		distanceToCurrentCentroid = CalculateDistance(dataX, centroidPosition,
				iExample, jCentroid);
		if (distanceToCurrentCentroid < smallestDistanceToCentroid) {
			smallestDistanceToCentroid = distanceToCurrentCentroid;
			assignedCentroid = jCentroid;
		}
	}

	assert(assignedCentroid != -1);
	return assignedCentroid;
}

void KmeansParallel::CompareTestResultsAgainstBaseline(
		float *centroidPosition) {
	int nCentroids = 3;
	float baseline[] = { 5.0059999999999993, 3.4180000000000006, 1.464,
			0.24399999999999991, 6.8538461538461526, 3.0769230769230766,
			5.7153846153846146, 2.0538461538461532, 5.8836065573770497,
			2.7409836065573772, 4.3885245901639349, 1.4344262295081966 };
	int i;
	float maxError = 1e-3;
	float error = 0;
	for (i = 0; i < nCentroids * nDim; i++)
		error += fabs(centroidPosition[i] - baseline[i]);

	assert(error < maxError);
	printf("OK!! Error agains baseline below threshold: %lf\n", error);
}

